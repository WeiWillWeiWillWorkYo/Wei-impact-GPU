#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "datatypes.cuh"
#include <fstream>
#include <string>
#include <iomanip> // 用于设置输出格式
#include "integration.cuh"
#include "VTKrelated.h"
#include "kernel.cuh"



int main() {
    const int numParticles = 1000;  // 粒子数量
    const double timeStep = 0.001;  // 时间步长
    const double G = 6.67430e-11;  // 万有引力常数
    //const double G = 6.67430e-3;
    const double viscosity = 0.01; //粘性系数
    const double gridCellSize = 0.2;  // 网格单元的大小
    const int numIterations = 100;  // 模拟总步数
    int3 gridDimensions = make_int3(30, 30, 30);  // 网格的三维尺寸
    
    Particles particles;
    hipError_t status = particles.allocate(numParticles);
    if (status != hipSuccess) {
        std::cerr << "CUDA error during particles allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    // 分配内存给交互对数组
    status = hipMallocManaged(&(particles.pairs), numParticles * sizeof(InteractionPair));
    if (status != hipSuccess) {
        std::cerr << "CUDA error during pairs allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    // 分配内存给交互对数量
    status = hipMallocManaged(&(particles.numPairs), sizeof(int));
    if (status != hipSuccess) {
        std::cerr << "CUDA error during numPairs allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    *particles.numPairs = 0;  // 初始化交互对数量


    Grid grid;
    status = grid.allocate(gridDimensions);
    if (status != hipSuccess) {
        std::cerr << "CUDA error during allocation: " << hipGetErrorString(status) << std::endl;
        particles.free();  // 释放粒子内存
        return 1;
    }

    int cubeRoot = cbrt(numParticles); // 计算粒子数量的立方根
    if (cubeRoot * cubeRoot * cubeRoot < numParticles) { // 确保有足够空间放置所有粒子
        cubeRoot++;
    }
    double spacing = 0.1; // 粒子间距
    int index = 0;

    // 以立方体形式初始化粒子
    for (int z = 0; z < cubeRoot; z++) {
        for (int y = 0; y < cubeRoot; y++) {
            for (int x = 0; x < cubeRoot; x++) {
                if (index < numParticles) {
                    // 位置和速度初始化
                    particles.x[index] = spacing * x;
                    particles.y[index] = spacing * y;
                    particles.z[index] = spacing * z;
                    particles.vx[index] = 0.0;
                    particles.vy[index] = 0.0;
                    particles.vz[index] = 0.0;
                    particles.mass[index] = 1000.0; // 假设所有粒子的质量为1

                    // 静态物理量初始化
                    particles.p[index] = 101325; // 假定初始压力为大气压
                    particles.rho[index] = 1000; // 假定初始密度，例如水的密度
                    particles.e[index] = 0; // 初始能量设为0
                    particles.c[index] = 343; // 假定初始声速，例如空气中的声速
                    particles.rho_prev[index] = 0; // 上一时刻的密度
                    particles.e_prev[index] = 0; // 上一时刻的能量
                    particles.h[index] = 0.1; // 光滑核半径

                    // 动态物理量初始化
                    particles.drhodt[index] = 0; // 密度变化率初始为0
                    particles.dedt[index] = 0; // 能量变化率初始为0
                    particles.av_x[index] = 0; // 平均速度初始化为0
                    particles.av_y[index] = 0;
                    particles.av_z[index] = 0;

                    // 张量初始化
                    for (int d = 0; d < 9; d++) {
                        particles.epsilon[index * 9 + d] = 0;
                        particles.tau[index * 9 + d] = 0;
                        particles.sigma[index * 9 + d] = 0;
                        particles.tau_prev[index * 9 + d] = 0;
                    }

                    // 其他初始化
                    particles.virt[index] = 0; // 假定所有粒子最初都不是虚拟粒子

                    index++;
                }
            }
        }
    }


    for (int i = 0; i < numParticles; i++) {
        std::cout << "Particle " << i << ": Position (" << particles.x[i] << ", " << particles.y[i] << ", " << particles.z[i] << ")\n";
    }

    int blockSize = 1;
    int numBlocks = (particles.count + blockSize - 1) / blockSize;

    // 为指针分配内存


    double h_avg = 0;
    for (int i = 0; i < numParticles; i++) {
        h_avg += particles.h[i];
    }
    h_avg /= numParticles;



    for (int iter = 0; iter < numIterations; iter++) {
        writeVTKFile(particles, iter);  // 输出VTK文件

        hipMemset(grid.sizes, 0, gridDimensions.x * gridDimensions.y * gridDimensions.z * sizeof(int));

        fillGrid <<<numBlocks, blockSize >>> (particles, grid, gridCellSize, gridDimensions);
        if (status != hipSuccess) {
            std::cerr << "CUDA error after HashFind: " << hipGetErrorString(status) << std::endl;
            break;
        }
        hipDeviceSynchronize();


        hashFind << <numBlocks, blockSize >> > (particles, grid, h_avg);

        hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after fillGrid: " << hipGetErrorString(status) << std::endl;
            break;
        }

        computeInteractions << <numBlocks, blockSize >> > (particles, timeStep, G);
        status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after computeInteractions: " << hipGetErrorString(status) << std::endl;
            break;
        }
        /**/
        updatePositions << <numBlocks, blockSize >> > (particles, timeStep);
        status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after updatePositions: " << hipGetErrorString(status) << std::endl;
            break;
        }
    }

    // 打印更新后的位置
    for (int i = 0; i < numParticles; i++) {
        std::cout << "Particle " << i << ": Position (" << particles.x[i] << ", " << particles.y[i] << ", " << particles.z[i] << ")\n";
    }
    std::cout << "Average search radius (h_avg): " << h_avg << std::endl;
    std::cout << "Particle spacing: " << spacing << std::endl;
    std::cout << "count: " << particles.count << std::endl;

    

    particles.free();  // 释放粒子内存
    grid.free();  // 释放网格内存



    return 0;
}