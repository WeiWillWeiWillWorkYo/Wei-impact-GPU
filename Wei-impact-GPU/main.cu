#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include "datatypes.cuh"
#include <fstream>
#include <string>
#include <iomanip> // 用于设置输出格式
#include "integration.cuh"
#include "VTKrelated.h"
#include "kernel.cuh"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>



int main() {
    const int numParticles = 100;  // 粒子数量
    const double timeStep = 0.001;  // 时间步长
    //const double G = 6.67430e-11;  // 万有引力常数
    const double G = 6.67430e-11;
    const double viscosity = 0.01; //粘性系数
    const double gridCellSize = 10;  // 网格单元的大小
    const int numIterations = 1000;  // 模拟总步数
    int3 gridDimensions = make_int3(30, 30, 30);  // 网格的三维尺寸
    
    Particles particles;
    hipError_t status = particles.allocate(numParticles);
    if (status != hipSuccess) {
        std::cerr << "CUDA error during particles allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    // 分配内存给交互对数组
    status = hipMallocManaged(&(particles.pairs), numParticles * sizeof(InteractionPair));
    if (status != hipSuccess) {
        std::cerr << "CUDA error during pairs allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    // 分配内存给交互对数量
    status = hipMallocManaged(&(particles.numPairs), sizeof(int));
    if (status != hipSuccess) {
        std::cerr << "CUDA error during numPairs allocation: " << hipGetErrorString(status) << std::endl;
        return 1;
    }

    *particles.numPairs = 0;  // 初始化交互对数量

    /*
    Grid grid;
    status = grid.allocate(gridDimensions);
    if (status != hipSuccess) {
        std::cerr << "CUDA error during allocation: " << hipGetErrorString(status) << std::endl;
        particles.free();  // 释放粒子内存
        return 1;
    }*/

    int cubeRoot = cbrt(numParticles); // 计算粒子数量的立方根
    if (cubeRoot * cubeRoot * cubeRoot < numParticles) { // 确保有足够空间放置所有粒子
        cubeRoot++;
    }
    double spacing = 0.1; // 粒子间距
    int index = 0;
    

    
    // 球体的半径和中心
    double radius = cubeRoot * spacing / 2.0;
    int3 centerA = make_int3(cubeRoot / 2, cubeRoot / 2, 3 * cubeRoot / 2); // 球A位于上方
    int3 centerB = make_int3(cubeRoot / 2, cubeRoot / 2, cubeRoot / 2);     // 球B位于下方

/*
    // 初始化粒子为两个球体
    for (int z = 0; z < 2 * cubeRoot; z++) {
        for (int y = 0; y < cubeRoot; y++) {
            for (int x = 0; x < cubeRoot; x++) {
                if (index < numParticles) {
                    double dxA = spacing * x - centerA.x;
                    double dyA = spacing * y - centerA.y;
                    double dzA = spacing * z - centerA.z;
                    double distanceA = sqrt(dxA * dxA + dyA * dyA + dzA * dzA);

                    double dxB = spacing * x - centerB.x;
                    double dyB = spacing * y - centerB.y;
                    double dzB = spacing * z - centerB.z;
                    double distanceB = sqrt(dxB * dxB + dyB * dyB + dzB * dzB);

                    if (distanceA <= radius || distanceB <= radius) {
                        // 确定粒子位置
                        particles.x[index] = spacing * x;
                        particles.y[index] = spacing * y;
                        particles.z[index] = spacing * z;

                        // 球B初始向上运动
                        if (distanceB <= radius) {
                            particles.vz[index] = 1.0; // 设定球B的初始速度向上
                        }
                        else {
                            particles.vz[index] = 0.0; // 球A静止
                        }

                        particles.vx[index] = 0.0;
                        particles.vy[index] = 0.0;

                        // 其余属性初始化相同
                        particles.mass[index] = 1000.0;
                        particles.p[index] = 101325;
                        particles.rho[index] = 1000;
                        particles.e[index] = 0;
                        particles.c[index] = 343;
                        particles.rho_prev[index] = 0;
                        particles.e_prev[index] = 0;
                        particles.h[index] = 0.5;
                        particles.drhodt[index] = 0;
                        particles.dedt[index] = 0;
                        particles.av_x[index] = 0;
                        particles.av_y[index] = 0;
                        particles.av_z[index] = 0;

                        for (int d = 0; d < 9; d++) {
                            particles.epsilon[index * 9 + d] = 0;
                            particles.tau[index * 9 + d] = 0;
                            particles.sigma[index * 9 + d] = 0;
                            particles.tau_prev[index * 9 + d] = 0;
                        }

                        particles.virt[index] = 0;

                        index++;
                    }
                }
            }
        }
    }
*/

    // 以立方体形式初始化粒子
    for (int z = 0; z < cubeRoot; z++) {
        for (int y = 0; y < cubeRoot; y++) {
            for (int x = 0; x < cubeRoot; x++) {
                if (index < numParticles) {
                    // 位置和速度初始化
                    particles.x[index] = spacing * x;
                    particles.y[index] = spacing * y;
                    particles.z[index] = spacing * z;
                    particles.vx[index] = 0.0;
                    particles.vy[index] = 0.0;
                    particles.vz[index] = 0.0;
                    particles.mass[index] = 1000.0; // 假设所有粒子的质量为1

                    // 静态物理量初始化
                    particles.p[index] = 101325; // 假定初始压力为大气压
                    particles.rho[index] = 1000; // 假定初始密度，例如水的密度
                    particles.e[index] = 0; // 初始能量设为0
                    particles.c[index] = 343; // 假定初始声速，例如空气中的声速
                    particles.rho_prev[index] = 0; // 上一时刻的密度
                    particles.e_prev[index] = 0; // 上一时刻的能量
                    particles.h[index] = 0.5; // 光滑核半径

                    // 动态物理量初始化
                    particles.drhodt[index] = 0; // 密度变化率初始为0
                    particles.dedt[index] = 0; // 能量变化率初始为0
                    particles.av_x[index] = 0; // 平均速度初始化为0
                    particles.av_y[index] = 0;
                    particles.av_z[index] = 0;

                    // 张量初始化
                    for (int d = 0; d < 9; d++) {
                        particles.epsilon[index * 9 + d] = 0;
                        particles.tau[index * 9 + d] = 0;
                        particles.sigma[index * 9 + d] = 0;
                        particles.tau_prev[index * 9 + d] = 0;
                    }

                    // 其他初始化
                    particles.virt[index] = 0; // 假定所有粒子最初都不是虚拟粒子

                    index++;
                }
            }
        }
    }
    particles.mass[42] = 10000000000000.0;

    for (int i = 0; i < numParticles; i++) {
        std::cout << "Particle " << i << ": Position (" << particles.x[i] << ", " << particles.y[i] << ", " << particles.z[i] << ")\n";
    }

    int blockSize = 4;
    int numBlocks = (particles.count + blockSize - 1) / blockSize;

    // 为指针分配内存


    double h_avg = 0;
    for (int i = 0; i < numParticles; i++) {
        h_avg += particles.h[i];
    }
    h_avg /= numParticles;

    //4.25 new iteration
    // 在迭代外部初始化
    thrust::device_vector<int> d_hashes(numParticles);
    thrust::device_vector<int> d_indices(numParticles);

    //粒子半径
    double particlerediu = 0.02;

    // 迭代循环
    for (int iter = 0; iter < numIterations; iter++) {
        writeVTKFile(particles, iter); // 输出VTK文件

        //hipMemset(grid.sizes, 0, gridDimensions.x * gridDimensions.y * gridDimensions.z * sizeof(int));

        thrust::copy(thrust::device, particles.hash, particles.hash + numParticles, d_hashes.begin());
        thrust::sequence(thrust::device, d_indices.begin(), d_indices.end());

        fillGrid << <numBlocks, blockSize >> > (particles, gridCellSize, gridDimensions);
        hipDeviceSynchronize();

        thrust::sort_by_key(d_hashes.begin(), d_hashes.end(), d_indices.begin());

        // 无需创建临时数组，直接在原数组上工作
        // 使用 permutation iterator 实现直接访问排序后的索引
        auto perm_x = thrust::make_permutation_iterator(particles.x, d_indices.begin());
        auto perm_y = thrust::make_permutation_iterator(particles.y, d_indices.begin());
        auto perm_z = thrust::make_permutation_iterator(particles.z, d_indices.begin());
        // 如有其他粒子属性，继续使用类似方式处理

        hashFind << <numBlocks, blockSize >> > (particles, numParticles);
        hipDeviceSynchronize();

        computeInteractions << <numBlocks, blockSize >> > (particles, timeStep, G);
        hipDeviceSynchronize();

        handleCollisions <<<numBlocks, blockSize >>> (particles, particlerediu);
        hipDeviceSynchronize();

        updatePositions << <numBlocks, blockSize >> > (particles, timeStep);
        hipDeviceSynchronize();
    }



/*
    for (int iter = 0; iter < numIterations; iter++) {
        writeVTKFile(particles, iter);  // 输出VTK文件

        hipMemset(grid.sizes, 0, gridDimensions.x * gridDimensions.y * gridDimensions.z * sizeof(int));


        fillGrid <<<numBlocks, blockSize >>> (particles, grid, gridCellSize, gridDimensions);
        if (status != hipSuccess) {
            std::cerr << "CUDA error after HashFind: " << hipGetErrorString(status) << std::endl;
            break;
        }
        hipDeviceSynchronize();

        hashFind << <numBlocks, blockSize >> > (particles, grid, h_avg);

        hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after fillGrid: " << hipGetErrorString(status) << std::endl;
            break;
        }

        computeInteractions << <numBlocks, blockSize >> > (particles, timeStep, G);
        status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after computeInteractions: " << hipGetErrorString(status) << std::endl;
            break;
        }

        updatePositions << <numBlocks, blockSize >> > (particles, timeStep);
        status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            std::cerr << "CUDA error after updatePositions: " << hipGetErrorString(status) << std::endl;
            break;
        }
    }
 */

    // 打印更新后的位置
    for (int i = 0; i < numParticles; i++) {
        std::cout << "Particle " << i << ": Position (" << particles.x[i] << ", " << particles.y[i] << ", " << particles.z[i] << ")\n";
        std::cout << "Particle" << i << ": acc (" << particles.ax[i] << ", " << particles.ay[i] << ", " << particles.az[i] << ")\n";
    }

    std::cout << "Average search radius (h_avg): " << h_avg << std::endl;
    std::cout << "Particle spacing: " << spacing << std::endl;
    std::cout << "count: " << particles.count << std::endl;

    

    particles.free();  // 释放粒子内存
    //grid.free();  // 释放网格内存



    return 0;
}