#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <math.h>
#include "kernel.cuh"
#include "constant.h"
#include "datatypes.cuh"

__device__ double cubicSplineKernel(double r, double h) {
    if (h <= 0) {
        printf("Error: h should be greater than 0.\n");
        return 0.0;
    }
    double q = r / h;
    double sigma = 1.0 / (PI * h * h * h);

    if (q <= 1.0) {
        if (q <= 0.5) {
            return sigma * (6.0 * (q * q * q - q * q) + 1.0);
        }
        else {
            return sigma * 2.0 * pow(1.0 - q, 3.0);
        }
    }
    else {
        return 0.0;
    }
}

__device__ double3 cubicSplineGradient(double3 r, double h) {
    if (h <= 0) {
        printf("Error: h should be greater than 0.\n");
        return make_double3(0.0, 0.0, 0.0);
    }
    double norm_r = sqrt(r.x * r.x + r.y * r.y + r.z * r.z);
    if (norm_r == 0.0) {
        printf("Warning: norm_r is 0.\n");
        return make_double3(0.0, 0.0, 0.0);
    }
    double q = norm_r / h;
    double sigma = 1.0 / (PI * h * h * h);
    double factor;

    if (q <= 1.0) {
        if (q <= 0.5) {
            factor = sigma * (18.0 * q - 12.0 * q * q) / h;
        }
        else {
            factor = sigma * -6.0 * (1.0 - q) * (1.0 - q) / h;
        }
        double3 grad = { r.x * factor / norm_r, r.y * factor / norm_r, r.z * factor / norm_r };
        return grad;
    }
    else {
        return make_double3(0.0, 0.0, 0.0);
    }
}

__device__ void kernel(double m_dist, double3 dist, double& weight, double3& grad, double h) {
    weight = cubicSplineKernel(m_dist, h);
    grad = cubicSplineGradient(dist, h);
    printf("Dist: %f, h: %f, Weight: %f, Grad: (%f, %f, %f)\n", m_dist, h, weight, grad.x, grad.y, grad.z);
}

