#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include "datatypes.cuh"
#include "integration.cuh"

__global__ void fillGrid(Particles p, Grid g, double gridCellSize, int3 gridDimensions) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < p.count) {
        // 偏移中心，以0为中心点
        int centerOffset = gridDimensions.x / 2;

        // 计算每个维度上的网格索引，加上中心偏移量
        int gridX = int(floor(p.x[i] / gridCellSize)) + centerOffset;
        int gridY = int(floor(p.y[i] / gridCellSize)) + centerOffset;
        int gridZ = int(floor(p.z[i] / gridCellSize)) + centerOffset;

        // 确保索引在有效范围内
        if (gridX >= 0 && gridX < gridDimensions.x &&
            gridY >= 0 && gridY < gridDimensions.y &&
            gridZ >= 0 && gridZ < gridDimensions.z) {
            int hash = gridX + gridY * gridDimensions.x + gridZ * gridDimensions.x * gridDimensions.y;
            p.hash[i] = hash;
            if (g.sizes[hash] < MAX_PARTICLES_PER_CELL) {
                int index = atomicAdd(&g.sizes[hash], 1);
                if (index < MAX_PARTICLES_PER_CELL) {
                    g.cells[hash * MAX_PARTICLES_PER_CELL + index] = i;
                    printf("Particle %d assigned to cell %d, position (%.2f, %.2f, %.2f), size now %d\n",
                        i, hash, p.x[i], p.y[i], p.z[i], g.sizes[hash]);
                }
            }
        }
    }
}
